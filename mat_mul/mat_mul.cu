#include "hip/hip_runtime.h"
#include "dinner123.h"
#include <cstring>
#include <cstdio>
#include <random>
#include <hip/hip_runtime.h>
#include <hipblas.h>

void mat_mul_cpu(const float *A, const float *B, size_t m, size_t n, size_t k, float *output) {
  Timer t;
  memset(output, 0, m * k * sizeof(float));
  for (int x = 0; x < m; x++) {
    for (int y = 0; y < n; y++) {
      for (int z = 0; z < k; z++) {
        output[x * k + z] += A[x * n + y] * B[y * k + z];
      }
    }
  }
  PrintTime();
}

__global__ void mat_mul_v1_kernel (const float *A, const float *B, size_t m, size_t n, size_t k, float *C) {
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;
  float sum = 0;
  for (int i = 0; i < n; i++) {
    sum = sum + A[x * n + i] * B[i * k + y];
  }
  C[x * k + y] = sum;
}

void mat_mul_v1(const float *A_h, const float *B_h, size_t m, size_t n, size_t k, float *C_h) {
  float *A_d, *B_d, *C_d;
  CHECK(hipMalloc(&A_d, m * n * sizeof(float)));
  CHECK(hipMalloc(&B_d, n * k * sizeof(float)));
  CHECK(hipMalloc(&C_d, m * k * sizeof(float)));
  CHECK(hipMemcpy(A_d, A_h, m * n * sizeof(float), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(B_d, B_h, n * k * sizeof(float), hipMemcpyHostToDevice));
  Timer t;
  dim3 block(32, 32);
  dim3 grid(m / block.x, k / block.y);
  mat_mul_v1_kernel<<<grid, block>>>(A_d, B_d, m, n, k, C_d);
  hipDeviceSynchronize();
  CHECK(hipGetLastError());
  PrintTime();
  CHECK(hipMemcpy(C_h, C_d, m * k * sizeof(float), hipMemcpyDeviceToHost));
  CHECK(hipFree(A_d));
  CHECK(hipFree(B_d));
  CHECK(hipFree(C_d));
}

#define BLOCK_SIZE 16

__global__ void mat_mul_v2_kernel (const float *A, const float *B, size_t m, size_t n, size_t k, float *C) {
  int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  int y = blockIdx.y * BLOCK_SIZE + threadIdx.y;
  float sum = 0;
  for (int i = 0; i < n / BLOCK_SIZE; i++) {
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
    As[threadIdx.x][threadIdx.y] = A[x * n + i * BLOCK_SIZE + threadIdx.y];
    Bs[threadIdx.x][threadIdx.y] = B[(i * BLOCK_SIZE + threadIdx.x) * k + y];
    __syncthreads();
    for (int j = 0; j < BLOCK_SIZE; j++) {
      sum += As[threadIdx.x][j] * Bs[j][threadIdx.y];
    }
    __syncthreads();
  }
  C[x * k + y] = sum;
}

void mat_mul_v2(const float *A_h, const float *B_h, size_t m, size_t n, size_t k, float *C_h) {
  float *A_d, *B_d, *C_d;
  CHECK(hipMalloc(&A_d, m * n * sizeof(float)));
  CHECK(hipMalloc(&B_d, n * k * sizeof(float)));
  CHECK(hipMalloc(&C_d, m * k * sizeof(float)));
  CHECK(hipMemcpy(A_d, A_h, m * n * sizeof(float), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(B_d, B_h, n * k * sizeof(float), hipMemcpyHostToDevice));
  Timer t;
  dim3 block(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid(m / BLOCK_SIZE, k / BLOCK_SIZE);
  mat_mul_v2_kernel<<<grid, block>>>(A_d, B_d, m, n, k, C_d);
  hipDeviceSynchronize();
  CHECK(hipGetLastError());
  PrintTime();
  CHECK(hipMemcpy(C_h, C_d, m * k * sizeof(float), hipMemcpyDeviceToHost));
  CHECK(hipFree(A_d));
  CHECK(hipFree(B_d));
  CHECK(hipFree(C_d));
}
#undef BLOCK_SIZE

void mat_mul_cub(const float *A_h, const float *B_h, size_t m, size_t n, size_t k, float *C_h) {
  float *A_d, *B_d, *C_d;
  CHECK(hipMalloc(&A_d, m * n * sizeof(float)));
  CHECK(hipMalloc(&B_d, n * k * sizeof(float)));
  CHECK(hipMalloc(&C_d, m * k * sizeof(float)));
  CHECK(hipMemcpy(A_d, A_h, m * n * sizeof(float), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(B_d, B_h, n * k * sizeof(float), hipMemcpyHostToDevice));
  Timer t;
  hipblasHandle_t handle;
  CUBLAS_CHECK(hipblasCreate(&handle));
  const float alpha = 1.0f;
  const float beta = 0.0f;
  CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                             k, m, n, &alpha, 
                             B_d, k, A_d, n, 
                             &beta, C_d, k));
  CHECK(hipDeviceSynchronize());
  PrintTime();
  CHECK(hipMemcpy(C_h, C_d, m * k * sizeof(float), hipMemcpyDeviceToHost));
  CHECK(hipFree(A_d));
  CHECK(hipFree(B_d));
  CHECK(hipFree(C_d));
}

#define BLOCK_SIZE 8
#define MAT_SIZE 8

__global__ void mat_mul_v3_kernel (const float4 *A, const float4 *B, size_t m, size_t n, size_t k, float *C) {
  int subx = threadIdx.x * MAT_SIZE;
  int suby = threadIdx.y * MAT_SIZE;
  int x = blockIdx.x * BLOCK_SIZE * MAT_SIZE + subx;
  int y = blockIdx.y * BLOCK_SIZE * MAT_SIZE + suby;
  float sum[MAT_SIZE][MAT_SIZE];
  for (int i = 0; i < MAT_SIZE; i++) {
    for (int j = 0; j < MAT_SIZE; j++) {
      sum[i][j] = 0;
    }
  }
  for (int i = 0; i < n / BLOCK_SIZE / MAT_SIZE; i++) {
    __shared__ float4 As[BLOCK_SIZE * MAT_SIZE][BLOCK_SIZE * MAT_SIZE / 4];
    __shared__ float4 Bs[BLOCK_SIZE * MAT_SIZE][BLOCK_SIZE * MAT_SIZE / 4];
    #pragma unroll
    for (int a = 0; a < MAT_SIZE; a++) {
      #pragma unroll
      for (int b = 0; b < MAT_SIZE; b += 4) {
        As[subx + a][(suby + b) >> 2] = A[((x + a) * n + i * BLOCK_SIZE * MAT_SIZE + suby + b) >> 2];
        Bs[suby + a][(subx + b) >> 2] = B[((y + a) * n + i * BLOCK_SIZE * MAT_SIZE + subx + b) >> 2];
      }
    }
    __syncthreads();
    float4 Ass, Bss;
    for (int j = 0; j < BLOCK_SIZE; j++) {
      #pragma unroll
      for (int a = 0; a < MAT_SIZE; a++) {
        #pragma unroll
        for (int b = 0; b < MAT_SIZE; b++) {
          #pragma unroll
          float s = 0;
          for (int c = 0; c < MAT_SIZE / 4; c++) {
            Ass = As[subx + a][j * MAT_SIZE / 4 + c];
            Bss = Bs[suby + b][j * MAT_SIZE / 4 + c];
            s += Ass.x * Bss.x + Ass.y * Bss.y + Ass.z * Bss.z + Ass.w * Bss.w;
          }
          sum[a][b] += s;
        }
      }
    }
    __syncthreads();
  }
  for (int a = 0; a < MAT_SIZE; a++) {
    for (int b = 0; b < MAT_SIZE; b++) {
      C[(x + a) * k + y + b] = sum[a][b];
    }
  }
}

void mat_mul_v3(const float *A_h, const float *B_h, size_t m, size_t n, size_t k, float *C_h) {
  float *B_T_h = (float*)malloc(n * k * sizeof(float));
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < k; j++) {
      B_T_h[j * n + i] = B_h[i * k + j];
    }
  }
  float4 *A_d, *B_d;
  float *C_d;
  CHECK(hipMalloc(&A_d, m * n * sizeof(float)));
  CHECK(hipMalloc(&B_d, n * k * sizeof(float)));
  CHECK(hipMalloc(&C_d, m * k * sizeof(float)));
  CHECK(hipMemcpy(B_d, B_T_h, n * k * sizeof(float), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(A_d, A_h, m * n * sizeof(float), hipMemcpyHostToDevice));
  Timer t;
  dim3 block(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid(m / BLOCK_SIZE / MAT_SIZE, k / BLOCK_SIZE / MAT_SIZE);
  mat_mul_v3_kernel<<<grid, block>>>(A_d, B_d, m, n, k, C_d);
  hipDeviceSynchronize();
  CHECK(hipGetLastError());
  PrintTime();
  CHECK(hipMemcpy(C_h, C_d, m * k * sizeof(float), hipMemcpyDeviceToHost));
  CHECK(hipFree(A_d));
  CHECK(hipFree(B_d));
  CHECK(hipFree(C_d));
  free(B_T_h);
}
#undef BLOCK_SIZE
#undef MAT_SIZE

signed main(){
  int m = 1 << 12, n = 1 << 12, k = 1 << 11;
  float *A, *B, *STD, *OUT;
  A = (float*)malloc(m * n * sizeof(float));
  B = (float*)malloc(n * k * sizeof(float));
  STD = (float*)malloc(m * k * sizeof(float));
  OUT = (float*)malloc(m * k * sizeof(float));
  
  initialData(A, m * n);
  initialData(B, n * k);
  mat_mul_cpu(A, B, m, n, k, STD);
  mat_mul_v1(A, B, m, n, k, OUT);
  checkResult(STD, OUT ,m * k);
  mat_mul_v2(A, B, m, n, k, OUT);
  checkResult(STD, OUT ,m * k);
  mat_mul_cub(A, B, m, n, k, OUT);
  checkResult(STD, OUT ,m * k);
  mat_mul_v3(A, B, m, n, k, OUT);
  checkResult(STD, OUT ,m * k);
  free(A);
  free(B);
  free(STD);
  free(OUT);
}